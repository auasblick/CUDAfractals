#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cuda_kernel.cuh"
#include <vector>

__global__ void vectorAdditionKernel(double* A, double* B, double* C, int arraySize) {
    // Get thread ID.
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if thread is within array bounds.
    if (threadID < arraySize) {
        // Add a and b.
        C[threadID] = A[threadID] + B[threadID];
    }
}



/**
 * Wrapper function for the CUDA kernel function.
 * @param A Array A.
 * @param B Array B.
 * @param C Sum of array elements A and B directly across.
 * @param arraySize Size of arrays A, B, and C.
 */
void cuda_sum(double* A, double* B, double* C, int arraySize) {

    // Initialize device pointers.
    double* d_A, * d_B, * d_C;

    // Allocate device memory.
    hipMalloc((void**)&d_A, arraySize * sizeof(double));
    hipMalloc((void**)&d_B, arraySize * sizeof(double));
    hipMalloc((void**)&d_C, arraySize * sizeof(double));

    // Transfer arrays a and b to device.
    hipMemcpy(d_A, A, arraySize * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, arraySize * sizeof(double), hipMemcpyHostToDevice);

    // Calculate blocksize and gridsize.
    dim3 blockSize(512, 1, 1);
    dim3 gridSize(512 / arraySize + 1, 1);

    // Launch CUDA kernel.
    vectorAdditionKernel <<<gridSize, blockSize >>> (d_A, d_B, d_C, arraySize);

    // Copy result array c back to host memory.
    hipMemcpy(C, d_C, arraySize * sizeof(double), hipMemcpyDeviceToHost);

}

void cuda_vecsum(std::vector<double>* A, std::vector<double>* B, std::vector<double>* C) {

    // Initialize device pointers.
    double* d_A, * d_B, * d_C;

    // Allocate device memory.
    hipMalloc((void**)&d_A, A->size() * sizeof(double));
    hipMalloc((void**)&d_B, A->size() * sizeof(double));
    hipMalloc((void**)&d_C, A->size() * sizeof(double));

    // Transfer arrays a and b to device.
    hipMemcpy(d_A, A, A->size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, A->size() * sizeof(double), hipMemcpyHostToDevice);

    // Calculate blocksize and gridsize.
    dim3 blockSize(512, 1, 1);
    dim3 gridSize(512 / A->size() + 1, 1);

    // Launch CUDA kernel.
    vectorAdditionKernel <<<gridSize, blockSize >>> (d_A, d_B, d_C, A->size());

    //temporary value
    //double* C_temp = C->data();

    // Copy result array c back to host memory.
    hipMemcpy(C, d_C, A->size() * sizeof(double), hipMemcpyDeviceToHost);
    //C->assign(C_temp, C_temp + C->size());

}